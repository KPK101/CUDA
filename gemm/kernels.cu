
#include <hip/hip_runtime.h>
/*
Author: Pranav Kumar Kota
year: 2024
*/

__global__ void naive_kernel(int M, int N, int K, float *A, float *B, float *C) {
    int row = blockIdx.y * blockDim.y + threadIdx.y;
    int col = blockIdx.x * blockDim.x + threadIdx.x;
    if (row < M && col < N) {
        float sum = 0;
        for (int i = 0; i < K; i++) {
            sum += A[row * K + i] * B[i * N + col];
        }
        C[row * N + col] = sum;
    }
}


__global__ void shared_kernel(int M, int N, int K, float *A, float *B, float *C){
    /*
    It is assumde that the block is square. To generalize validity, add additional checks (TBD)
    */

    int globalRow = blockIdx.y*blockDim.y + threadIdx.y;
    int globalCol = blockIdx.x*blockDim.x + threadIdx.x;

    int localRow = threadIdx.y;
    int localCol = threadIdx.x;

    // define shared memory to load tiles of input matrices A & B
    extern __shared__ float sm[];
    float *ATile = sm;
    float *BTile = sm + blockDim.x*blockDim.y;

    // thread sum for output index
    float sum = 0.0f;
    int phases = (K + blockDim.x - 1)/blockDim.x;

    if(globalRow==0 && globalCol==0){
        printf("Phases: %d\n", phases);
    }
    for(int p=0; p<phases; p++){
        __syncthreads();
        // load A into shared memory
        if(globalRow<M && p*blockDim.x + localCol < K){
            ATile[localRow*blockDim.x + localCol] = A[globalRow*K + p*blockDim.x + localCol];
        }
        else{
            ATile[localRow*blockDim.x + localCol] = 0.0f;
        }
        // load B into shared memory
        if(p*blockDim.y + localRow < K && globalCol < N){
            BTile[localRow*blockDim.x + localCol] = B[(p*blockDim.y + localRow)*N + globalCol];
        }
        else{
            BTile[localRow*blockDim.x + localCol] = 0.0f;
        }
        // sync to wait for all data to be loaded
        __syncthreads();
        // perform matrix multiplication between loaded tiles and update sum
        for(int j=0; j<blockDim.x; j++){
            sum += ATile[localRow*blockDim.x + j]*BTile[j*blockDim.x + localCol];
        }
    }

    if(globalRow<M && globalCol<N){
        C[globalRow*N + globalCol] = sum;
    }
}
